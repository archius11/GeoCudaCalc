#include "hip/hip_runtime.h"

//#define ITEM_COUNT 2
#define _PI 3.14159265358979323846
#define _PI2 1.57079632679489661923
#define _RAD 6372795



#include "hip/hip_runtime.h"
#include ""

#include <math.h>
#include <iostream>
#include <stdio.h>
#include <string>

using namespace std;

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);


__global__ void geo_invert(double2* d_dot1, double2* d_dot2, double* d_dist, double* d_azimut, long count)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < count)
	{
		d_dot1[idx].x = d_dot1[idx].x * _PI / 180;	//lat1
		d_dot1[idx].y = d_dot1[idx].y * _PI / 180;	//lng1
		d_dot2[idx].x = d_dot2[idx].x * _PI / 180;	//lat2
		d_dot2[idx].y = d_dot2[idx].y * _PI / 180;	//lng2

		double cl1, cl2, sl1, sl2, delta, cdelta, sdelta;
		cl1 = cos(d_dot1[idx].x);
		cl2 = cos(d_dot2[idx].x);
		sl1 = sin(d_dot1[idx].x);
		sl2 = sin(d_dot2[idx].x);
		delta = d_dot2[idx].y - d_dot1[idx].y;
		cdelta = cos(delta);
		sdelta = sin(delta);

		double x, y, z, ad, z2;
		y = sqrt(pow(cl2*sdelta, 2) + pow(cl1*sl2 - sl1*cl2*cdelta, 2));
		x = sl1*sl2 + cl1*cl2*cdelta;
		ad = atan(y / x);
		d_dist[idx] = ad * _RAD;

		x = (cl1*sl2) - (sl1*cl2*cdelta);
		y = sdelta*cl2;

		if (x == 0)
		{
			if (y > 0)
				z = -90;
			else if (y < 0)
				z = 90;
			else if (y == 0)
				z = 0;
		}
		else
		{
			z = atan(-y / x) * 180 / _PI;
			if (x < 0)
			{
				z = z + 180;
			}
		}

		z2 = z + 180.0f;

		while (z2 >= 360)
		{
			z2 = z2 - 360;
		}

		z2 = z2 - 180;


		z2 = -z2 * _PI / 180;
		double anglerad2;
		anglerad2 = z2 - ((2 * _PI) * floor(z2 / (2 * _PI)));
		d_azimut[idx] = anglerad2 * 180 / _PI;


	}
}

__device__ double CartToSpher(double3* x, double2* y)
{
	double p;	

	p = hypot(x->x, x->y); //0,566796731779912 (0,566796731779913) 0,000000000000001
	y->y = atan(x->y / x->x); //0,658744870833875 (0,658744870833875) 0
	y->x = atan(x->z / p); //0,968183828701654 (0,968183828701654) 0

	return hypot(p, x->z);
}

__device__ void SpherToCart(double2* y, double3* x)
{
	double p;

	p = cos(y->x); //0,509535037739044 (0,509535037739044) 0
	x->z = sin(y->x); //0,860449908661899 (0,860449908661899)  0
	x->y = p * sin(y->y); //sin -0,896141078848377 () ; -0,456615278430515379561368931588 (-0,456615278430516) -0,0000000000000007
	x->x = p * cos(y->y); //cos -0,443769272032738 () ; -0,226115992772629239921468822472 (-0,226115992772629)  0,0000000000000002

	return;
}

__device__ void Rotate2(double3* x, double a)
{
	double c, s, xj;

	c = cos(a); //-0,0246449569641315 (-0,0246449569641315) 0
	s = sin(a); //-0,999696266921226 (-0,999696266921226) 0
	xj = x->x * c + x->y * s; //0,448200835371488472286581966712384075135922396544509757722076 (0,448200835371489) 0,0000000000000006
	x->y = -x->x * s + x->y * c; //0,346950351388623496054477853457869740953054474470049783262896 (0,346950351388624) 0,0000000000000006
	x->x = xj; //0,448200835371488472286581966712384075135922396544509757722076 (0,448200835371489) 0,0000000000000006

	return;
}

__device__ void Rotate1(double3* x, double a)
{
	double c, s, xj;

	c = cos(a); //0,799692643650457 (0,799692643650457) 0
	s = sin(a); //-0,60040958993952 (-0,60040958993952) 0
	xj = x->z * c + x->x * s; //0,82385767268601006086792194931074587209689344 (0,823857672686011) 0,000000000000001
	x->x = -x->z * s + x->x * c; //0,335799080791196711492751735883669680645330296 (0,335799080791197) 0,0000000000000003
	x->z = xj; //0,82385767268601006086792194931074587209689344 (0,823857672686011) 0,000000000000001

	return;
}

__device__ void SphereDirect(double2* pt1, double azi, double dist, double2* pt2)
{
	double2 pt;
	double3 x;

	pt.x = _PI2 - dist; //1,036151994127676035482994172886465044 (1,03615199412768)	0,000000000000004 15
	pt.y = _PI - azi;  // -2,030596755794815247590950216888888888888888888888888889 (-2,03059675579482)  -0,000000000000005 15

	SpherToCart(&pt, &x);               // ����������� -> ���������
	Rotate1(&x, pt1->x - _PI2); // ������ ��������
	Rotate2(&x, -pt1->y);           // ������ ��������
	CartToSpher(&x, pt2);           // ��������� -> ����������� 

}

__global__ void geo_direct(double2* d_dot1, double* d_dist, double* d_azimut, double2* d_dot2, long count)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < count)
	{
		d_dot1[idx].x = d_dot1[idx].x * _PI / 180;	//pt1[0] 0,926783132217889 (0.926783132217889) 0
		d_dot1[idx].y = d_dot1[idx].y * _PI / 180;	//pt1[1] 1,595443779225097 (1,5954437792251) 0,000000000000003 15
													//d_azimut[idx] 296,344624 (296.344624)
		d_azimut[idx] = d_azimut[idx] * _PI / 180; //5,172189409384608486050950216 (5,17218940938461) 0,000000000000002 15
		d_dist[idx] = d_dist[idx] / _RAD; //0,534644332667220583747005827113534956 (0,534644332667221)	0,0000000000000005 15

		double2 pt2;

		SphereDirect(&d_dot1[idx], d_azimut[idx], d_dist[idx], &pt2);

		d_dot2[idx].x = pt2.x * 180 / _PI; //pt 0,968183828701654 (0,968183734421639) // 55,472847177421830384141692371164405517504840227 (55,4728417755749) // -0,000000094280015 // -0,00000540184693
		d_dot2[idx].y = pt2.y * 180 / _PI; //pt 0,658744870833875 (0,658744865257753) // 37,743300874671594906787964611067062014639516391 (37,7433005551833) // -0,000000005576122 // -0,00000031948829

		if (d_dot2[idx].x < 0)
			d_dot2[idx].x += 180;

		if (d_dot2[idx].y < 0)
			d_dot2[idx].y += 180;

	}
}

/*__global__ void geo_invert(double2* d_dot1, double2* d_dot2, double* d_dist, double* d_azimut, long count)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < count)
	{
		double3 x;
		double2 pt;

		d_dot1[idx].x = d_dot1[idx].x * _PI / 180;	//lat1
		d_dot1[idx].y = d_dot1[idx].y * _PI / 180;	//lng1
		d_dot2[idx].x = d_dot2[idx].x * _PI / 180;	//lat2
		d_dot2[idx].y = d_dot2[idx].y * _PI / 180;	//lng2


		SpherToCart(&d_dot2[idx], &x);			// ����������� -> ���������
		Rotate2(&x, d_dot1[idx].y);			// ������ ��������
		Rotate1(&x, _PI2 - d_dot1[idx].x);	// ������ ��������
		CartToSpher(&x, &pt);	     		// ��������� -> �����������
		d_azimut[idx] = (_PI - pt.y)* 180 / _PI;
		d_dist[idx] = (_PI2 - pt.x) * _RAD;
	}
}*/

__device__ double d_abs(double var)
{
	if (var<0)
	{
		return -var;
	}
	else
	{
		return var;
	}
}

__global__ void d_cudainit(int *a, int *b)
{
    int i = threadIdx.x;
	if (i==1)
	{
		b[i] = a[i] * 2;
	}
}

__device__ void SpherToCartR(double* y, double* x)
{
	double y_lat = y[0] * _PI / 180;
	double y_lon = y[1] * _PI / 180;

	double cos_y_lat = cos(y_lat);

	x[0] = _RAD * cos_y_lat * cos(y_lon);
	x[1] = _RAD * cos_y_lat * sin(y_lon);
	x[2] = _RAD * sin(y_lat);

	return;
}

__device__ bool LineIsTooFar(double* M, double* A, double* B, float max_delta)
{
	double lat_lag = 0.00001 * max_delta * 1.2;
	double lon_lag = 0.00003 * max_delta * 1.2;

	double sqr_left_low_lat = M[0] - lat_lag;
	double sqr_left_low_lon = M[1] - lon_lag;
	double sqr_right_high_lat = M[0] + lat_lag;
	double sqr_right_high_lon = M[1] + lon_lag;

	if (A[1] < sqr_left_low_lon && B[1] < sqr_left_low_lon)
		return true;

	if (A[1] > sqr_right_high_lon && B[1] > sqr_right_high_lon)
		return true;

	if (A[0] < sqr_left_low_lat && B[0] < sqr_left_low_lat)
		return true;

	if (A[0] > sqr_right_high_lat && B[0] > sqr_right_high_lat)
		return true;

	return false;
}

__device__ double LineLength(double* a, double* b)
{
	double K1, K2, K3, K4;

	K1 = b[0] * a[0] + b[1] * a[1] + b[2] * a[2];
	K2 = sqrt(pow(b[0], 2) + pow(b[1], 2) + pow(b[2], 2));
	K3 = sqrt(pow(a[0], 2) + pow(a[1], 2) + pow(a[2], 2));
	K4 = K1 / (K2 * K3);

	if (K4 > 1) K4 = 1;

	return _RAD * acos(K4);
}

__device__ bool M_ProjectionOnPlane(double* m_dec, double plane_A, double plane_B, double plane_C, double* a_dec, double* b_dec)
{
	double t = -(plane_A * m_dec[0] + plane_B * m_dec[1] + plane_C * m_dec[2]) / (pow(plane_A, 2) + pow(plane_B, 2) + pow(plane_C, 2));

	double* dot_on_plane = new double[3];
	dot_on_plane[0] = plane_A * t + m_dec[0];
	dot_on_plane[1] = plane_B * t + m_dec[1];
	dot_on_plane[2] = plane_C * t + m_dec[2];

	double* dot_k = new double[3];
	double K = sqrt(pow(dot_on_plane[0], 2) + pow(dot_on_plane[1], 2) + pow(dot_on_plane[2], 2));
	dot_k[0] = (_RAD * dot_on_plane[0]) / K;
	dot_k[1] = (_RAD * dot_on_plane[1]) / K;
	dot_k[2] = (_RAD * dot_on_plane[2]) / K;

	double line_AB = LineLength(a_dec, b_dec);
	double line_AK = LineLength(a_dec, dot_k);
	double line_BK = LineLength(b_dec, dot_k);

	delete dot_on_plane;
	delete dot_k;

	return d_abs(line_AK + line_BK - line_AB) < 0.01;
}

__device__ double DistanceToLine(double* m_dec, double* a_dec, double* b_dec, bool calc_mb)
{
	if (a_dec[0] == b_dec[0] &&
		a_dec[1] == b_dec[1] &&
		a_dec[2] == b_dec[2])
	{
		return LineLength(m_dec, a_dec);
	}

	double plane_A, plane_B, plane_C;
	plane_A = a_dec[1] * b_dec[2] - a_dec[2] * b_dec[1];
	plane_B = a_dec[2] * b_dec[0] - a_dec[0] * b_dec[2];
	plane_C = a_dec[0] * b_dec[1] - a_dec[1] * b_dec[0];

	double d, MK_length, MA_length, MB_length, minlength;

	if (M_ProjectionOnPlane(m_dec, plane_A, plane_B, plane_C, a_dec, b_dec))
	{
		d = d_abs(plane_A*m_dec[0] + plane_B * m_dec[1] + plane_C*m_dec[2]) / sqrt(pow(plane_A, 2) + pow(plane_B, 2) + pow(plane_C, 2));
		MK_length = _RAD * asin(d / _RAD);
		minlength = MK_length;
	}
	else
	{
		MA_length = LineLength(m_dec, a_dec);
		if (calc_mb)
		{
			MB_length = LineLength(m_dec, b_dec);
			minlength = fmin(MA_length, MB_length);
		}
		else
		{
			minlength = MA_length;
		}
	}

	return minlength;
}

__device__ int dot_near_polyline(double dot_lat, double dot_lon, double* line_lat, double* line_lon, long line_count, float max_delta)
{
	double* m_sph = new double[2];
	m_sph[0] = dot_lat;
	m_sph[1] = dot_lon;

	double* m_dec = new double[3];

	SpherToCartR(m_sph, m_dec);

	double disttoline;
	int dotisclose = 0;

	double* a_sph = new double[2];
	double* b_sph = new double[2];
	double* a_dec = new double[3];
	double* b_dec = new double[3];

	for (long i = 0; i <= line_count - 2; i++)
	{
		//line dots
		a_sph[0] = line_lat[i];
		a_sph[1] = line_lon[i];

		b_sph[0] = line_lat[i+1];
		b_sph[1] = line_lon[i+1];

		if (LineIsTooFar(m_sph, a_sph, b_sph, max_delta))
		{
			continue;
		}

		SpherToCartR(a_sph, a_dec);
		SpherToCartR(b_sph, b_dec);

		disttoline = DistanceToLine(m_dec, a_dec, b_dec, i == line_count - 2);

		if (disttoline < max_delta)
		{
			dotisclose = 1;
			break;
		}
	}

	delete m_sph;
	delete m_dec;
	delete a_sph;
	delete b_sph;
	delete a_dec;
	delete b_dec;

	return dotisclose;



}

__global__ void dotarray_near_polyline(double* dot_lat, double* dot_lon, double* line_lat, double* line_lon, long dot_count, long line_count, float max_delta, int* dot_result)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < dot_count)
	{
		dot_result[idx] = dot_near_polyline(dot_lat[idx], dot_lon[idx], line_lat, line_lon, line_count, max_delta);
		//one thread calculate 1 dot near line array
	}
}


extern "C" __declspec(dllexport)	int GetInvertGeo(double* dot1_lat, double* dot1_lon, double* dot2_lat, double* dot2_lon, double* dist, double* azimut, long count)
{
	const int blockSize = 1024;
	int numOfBlocks = (count + blockSize - 1) / blockSize;
	dim3 dimGrid(numOfBlocks);
	dim3 dimBlock(blockSize);
	hipError_t cudaStatus;

	int size_double = sizeof(double) * count;
	int size_double2 = sizeof(double2) * count;

	double2 *d_dot1, *d_dot2;
	double *d_azimut;
	double *d_dist;
	cudaStatus = hipMalloc((void**)&d_dot1, size_double2);
	if (cudaStatus != hipSuccess)
	{
		return 2;
	}
	cudaStatus = hipMalloc((void**)&d_dot2, size_double2);
	if (cudaStatus != hipSuccess)
	{
		return 2;
	}
	cudaStatus = hipMalloc((void**)&d_azimut, size_double);
	if (cudaStatus != hipSuccess)
	{
		return 2;
	}
	cudaStatus = hipMalloc((void**)&d_dist, size_double);
	if (cudaStatus != hipSuccess)
	{
		return 2;
	}

	double2 *dot1 = new double2[count];
	double2 *dot2 = new double2[count];

	for (long i = 0; i < count; i++)
	{
		dot1[i].x = dot1_lat[i];
		dot1[i].y = dot1_lon[i];
		dot2[i].x = dot2_lat[i];
		dot2[i].y = dot2_lon[i];
	}


	//��������� ������� ������� � �����������
	cudaStatus = hipMemcpy(d_dot1, dot1, size_double2, hipMemcpyKind::hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		return 3;
	}
	cudaStatus = hipMemcpy(d_dot2, dot2, size_double2, hipMemcpyKind::hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		return 3;
	}

	delete dot1;
	delete dot2;

	geo_invert <<< dimGrid, dimBlock >>> (d_dot1, d_dot2, d_dist, d_azimut, count);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		return 4;
	}

	//double *dist = new double[ITEM_COUNT];
	//double *azimut = new double[ITEM_COUNT];
	cudaStatus = hipMemcpy(azimut, d_azimut, size_double, hipMemcpyKind::hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
	{
		return 3;
	}
	cudaStatus = hipMemcpy(dist, d_dist, size_double, hipMemcpyKind::hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
	{
		return 3;
	}


	cudaStatus = hipFree(d_dot1);
	if (cudaStatus != hipSuccess)
	{
		return 5;
	}
	cudaStatus = hipFree(d_dot2);
	if (cudaStatus != hipSuccess)
	{
		return 5;
	}
	cudaStatus = hipFree(d_azimut);
	if (cudaStatus != hipSuccess)
	{
		return 5;
	}
	cudaStatus = hipFree(d_dist);
	if (cudaStatus != hipSuccess)
	{
		return 5;
	}

	//����� �����������
	//cout.precision(6);
	//cout << dist[0] << endl << azimut[0] << endl;

	//delete dist;
	//delete azimut;
	return 0;
}

extern "C" __declspec(dllexport)	int GetDirectGeo(double* dot1_lat, double* dot1_lon, double* dist, double* azimut,  double* dot2_lat, double* dot2_lon, long count)
{
	const int blockSize = 1024;
	//int numOfBlocks = (ITEM_COUNT + blockSize - 1) / blockSize;
	int numOfBlocks = (count - 1) / blockSize + 1;
	dim3 dimGrid(numOfBlocks);
	dim3 dimBlock(blockSize);
	hipError_t cudaStatus;

	int size_double = sizeof(double) * count;
	int size_double2 = sizeof(double2) * count;

	double2 *d_dot1, *d_dot2;
	double *d_azimut;
	double *d_dist;
	cudaStatus = hipMalloc((void**)&d_dot1, size_double2);
	if (cudaStatus != hipSuccess)
	{
		return 2;
	}
	cudaStatus = hipMalloc((void**)&d_dot2, size_double2);
	if (cudaStatus != hipSuccess)
	{
		return 2;
	}
	cudaStatus = hipMalloc((void**)&d_azimut, size_double);
	if (cudaStatus != hipSuccess)
	{
		return 2;
	}
	cudaStatus = hipMalloc((void**)&d_dist, size_double);
	if (cudaStatus != hipSuccess)
	{
		return 2;
	}


	//������� ��� �����
	//double2 *dot1 = new double2[ITEM_COUNT];
	//double *azimut = new double[ITEM_COUNT];
	//double *dist = new double[ITEM_COUNT];

	//��������� �����
	//dot1[0].x = 53.100762;
	//dot1[0].y = 91.412195;
	//azimut[0] = 296.344624;
	//dist[0] = 3407178.73;


	double2 *dot1 = new double2[count];
	double2 *dot2 = new double2[count];

	for (long i = 0; i < count; i++)
	{
		dot1[i].x = dot1_lat[i];
		dot1[i].y = dot1_lon[i];
	}

	cudaStatus = hipMemcpy(d_dot1, dot1, size_double2, hipMemcpyKind::hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		return 3;
	}
	cudaStatus = hipMemcpy(d_azimut, azimut, size_double, hipMemcpyKind::hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		return 3;
	}
	cudaStatus = hipMemcpy(d_dist, dist, size_double, hipMemcpyKind::hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		return 3;
	}

	delete dot1;

	geo_direct <<< dimGrid, dimBlock >>> (d_dot1, d_dist, d_azimut,  d_dot2, count);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		return 4;
	}


	//double2 *dot2 = new double2[ITEM_COUNT];
	cudaStatus = hipMemcpy(dot2, d_dot2, size_double2, hipMemcpyKind::hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
	{
		return 3;
	}

	for (long i = 0; i < count; i++)
	{
		dot2_lat[i] = dot2[i].x;
		dot2_lon[i] = dot2[i].y;
	}

	delete dot2;

	cudaStatus = hipFree(d_dot1);
	if (cudaStatus != hipSuccess)
	{
		return 5;
	}
	cudaStatus = hipFree(d_dot2);
	if (cudaStatus != hipSuccess)
	{
		return 5;
	}
	cudaStatus = hipFree(d_azimut);
	if (cudaStatus != hipSuccess)
	{
		return 5;
	}
	cudaStatus = hipFree(d_dist);
	if (cudaStatus != hipSuccess)
	{
		return 5;
	}

	//����� �����������
	//cout.precision(10);
	//cout << dot2[0].x << endl << dot2[0].y << endl;

	//delete dist;
	//delete azimut;
	return 0;
}

extern "C" __declspec(dllexport)	int CudaInitialize()
{
    int* a = new int[1];
    int* b = new int[1];
	hipError_t cudaStatus;

	const int blockSize = 1024;
	int numOfBlocks = 1;
	dim3 dimGrid(numOfBlocks);
	dim3 dimBlock(blockSize);

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess)
	{
		return 1;
	}

	int *d_a;
	int *d_b;
	cudaStatus = hipMalloc((void**)&d_a, sizeof(int));
	if (cudaStatus != hipSuccess)
	{
		return 2;
	}
	cudaStatus = hipMalloc((void**)&d_b, sizeof(int));
	if (cudaStatus != hipSuccess)
	{
		return 2;
	}

	a[0] = 1;
	
	cudaStatus = hipMemcpy(d_a, a, sizeof(int), hipMemcpyKind::hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		return 3;
	}

	d_cudainit <<< dimGrid, dimBlock >>> (d_a, d_b);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		return 4;
	}

	cudaStatus = hipMemcpy(b, d_b, sizeof(int), hipMemcpyKind::hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
	{
		return 3;
	}

	cudaStatus = hipFree(d_a);
	if (cudaStatus != hipSuccess)
	{
		return 5;
	}
	cudaStatus = hipFree(d_b);
	if (cudaStatus != hipSuccess)
	{
		return 5;
	}

	delete a;
	delete b;

    return 0;
}

extern "C" __declspec(dllexport)	int DotArrayNearPolyline(double* dot_lat, double* dot_lon, double* line_lat, double* line_lon, long dot_count, long line_count, float max_delta, int* dot_result, char str[])
{
	const int blockSize = 1024;
	//int numOfBlocks = (ITEM_COUNT + blockSize - 1) / blockSize;
	int numOfBlocks = (dot_count - 1) / blockSize + 1;
	dim3 dimGrid(numOfBlocks);
	dim3 dimBlock(blockSize);
	hipError_t cudaStatus;

	int size_double_dots = sizeof(double) * dot_count;
	int size_double_polyline = sizeof(double) * line_count;
	int size_int = sizeof(int) * dot_count;

	double *d_dot_lat;
	double *d_dot_lon;
	double *d_line_lat;
	double *d_line_lon;
	int *d_dot_result;

	cudaStatus = hipMalloc((void**)&d_dot_lat, size_double_dots);
	if (cudaStatus != hipSuccess)
	{
		return 2;
	}
	cudaStatus = hipMalloc((void**)&d_dot_lon, size_double_dots);
	if (cudaStatus != hipSuccess)
	{
		return 2;
	}
	cudaStatus = hipMalloc((void**)&d_line_lat, size_double_polyline);
	if (cudaStatus != hipSuccess)
	{
		return 2;
	}
	cudaStatus = hipMalloc((void**)&d_line_lon, size_double_polyline);
	if (cudaStatus != hipSuccess)
	{
		return 2;
	}
	cudaStatus = hipMalloc((void**)&d_dot_result, size_int);
	if (cudaStatus != hipSuccess)
	{
		return 2;
	}

	cudaStatus = hipMemcpy(d_dot_lat, dot_lat, size_double_dots, hipMemcpyKind::hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		return 3;
	}
	cudaStatus = hipMemcpy(d_dot_lon, dot_lon, size_double_dots, hipMemcpyKind::hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		return 3;
	}
	cudaStatus = hipMemcpy(d_line_lat, line_lat, size_double_polyline, hipMemcpyKind::hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		return 3;
	}
	cudaStatus = hipMemcpy(d_line_lon, line_lon, size_double_polyline, hipMemcpyKind::hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		return 3;
	}


	dotarray_near_polyline <<< dimGrid, dimBlock >>> (d_dot_lat, d_dot_lon, d_line_lat, d_line_lon, dot_count, line_count, max_delta, d_dot_result);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		const char* cudaerr = hipGetErrorString(cudaStatus);
		//char source[] = (char[])cudaerr;
		int Size;
		while (cudaerr[Size] != '\0') Size++;
		sprintf_s(str, Size, cudaerr);
		return 4;
	}

	cudaStatus = hipMemcpy(dot_result, d_dot_result, size_int, hipMemcpyKind::hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
	{
		return 3;
	}

	cudaStatus = hipFree(d_dot_lat);
	if (cudaStatus != hipSuccess)
	{
		return 5;
	}
	cudaStatus = hipFree(d_dot_lon);
	if (cudaStatus != hipSuccess)
	{
		return 5;
	}	
	cudaStatus = hipFree(d_line_lat);
	if (cudaStatus != hipSuccess)
	{
		return 5;
	}	
	cudaStatus = hipFree(d_line_lon);
	if (cudaStatus != hipSuccess)
	{
		return 5;
	}	
	cudaStatus = hipFree(d_dot_result);
	if (cudaStatus != hipSuccess)
	{
		return 5;
	}

	return 0;
}
